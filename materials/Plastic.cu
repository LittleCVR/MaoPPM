#include "hip/hip_runtime.h"
/*
 * =============================================================================
 *
 *       Filename:  Plastic.cu
 *
 *    Description:  
 *
 *        Version:  1.0
 *        Created:  2011-06-21 12:59:55
 *
 *         Author:  Chun-Wei Huang (LittleCVR), 
 *        Company:  Communication & Multimedia Laboratory,
 *                  Department of Computer Science & Information Engineering,
 *                  National Taiwan University
 *
 * =============================================================================
 */

#include    "Plastic.h"

/*----------------------------------------------------------------------------
 *  Header files from OptiX
 *----------------------------------------------------------------------------*/
#include    <optix_world.h>

/*----------------------------------------------------------------------------
 *  header files of our own
 *----------------------------------------------------------------------------*/
#include    "global.h"
#include    "payload.h"
#include    "utility.h"
#include    "BSDF.h"
#include    "DifferentialGeometry.h"
#include    "Intersection.h"

/*----------------------------------------------------------------------------
 *  namespace
 *----------------------------------------------------------------------------*/
using namespace optix;
using namespace MaoPPM;



rtDeclareVariable(uint2, launchIndex, rtLaunchIndex, );
rtDeclareVariable(uint2, launchSize ,              , );

rtBuffer<float,  1>  sampleList;
rtBuffer<char,   1>  inputHeap;

rtDeclareVariable(rtObject, rootObject, , );
rtDeclareVariable(DifferentialGeometry, geometricDG, attribute differential_geometry, ); 

rtDeclareVariable(float , rayEpsilon ,                       , );
rtDeclareVariable(Ray   , currentRay , rtCurrentRay          , );
rtDeclareVariable(float , tHit       , rtIntersectionDistance, );
rtDeclareVariable(NormalRayPayload, normalRayPayload, rtPayload, );
rtDeclareVariable(ShadowRayPayload, shadowRayPayload, rtPayload, );

rtDeclareVariable(Index, materialIndex, , );



/* 
 * ===  FUNCTION  ==============================================================
 *         Name:  handleNormalRayClosestHit
 *  Description:  
 * =============================================================================
 */
RT_PROGRAM void handleNormalRayClosestHit()
{
    normalRayPayload.isHit = true;
    // Intersection
    Intersection * intersection = normalRayPayload.intersection();
    // Differential geometry.
    *(intersection->dg()) = geometricDG;
    // Material
    intersection->m_material = LOCAL_HEAP_GET_OBJECT_POINTER(Material, materialIndex);
}   /* -----  end of function handleNormalRayClosestHit  ----- */
