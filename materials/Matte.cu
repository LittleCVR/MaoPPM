#include "hip/hip_runtime.h"
/*
 * =============================================================================
 *
 *       Filename:  Matte.cu
 *
 *    Description:  Matte material device codes.
 *
 *        Version:  1.0
 *        Created:  2011-06-21 12:59:55
 *
 *         Author:  Chun-Wei Huang (LittleCVR), 
 *        Company:  Communication & Multimedia Laboratory,
 *                  Department of Computer Science & Information Engineering,
 *                  National Taiwan University
 *
 * =============================================================================
 */

#include    "Matte.h"

/*----------------------------------------------------------------------------
 *  Header files from OptiX
 *----------------------------------------------------------------------------*/
#include    <optix_world.h>

/*----------------------------------------------------------------------------
 *  header files of our own
 *----------------------------------------------------------------------------*/
#include    "global.h"
#include    "payload.h"
#include    "reflection.h"
#include    "utility.h"
#include    "DifferentialGeometry.h"
#include    "Intersection.h"

/*----------------------------------------------------------------------------
 *  namespace
 *----------------------------------------------------------------------------*/
using namespace optix;
using namespace MaoPPM;



rtDeclareVariable(uint2, launchIndex, rtLaunchIndex, );
rtDeclareVariable(uint2, launchSize ,              , );

rtBuffer<char,  1>  heap;
rtBuffer<float, 1>  sampleList;

rtDeclareVariable(rtObject, rootObject, , );
rtDeclareVariable(DifferentialGeometry, geometricDG, attribute differential_geometry, ); 

rtDeclareVariable(float , rayEpsilon ,                       , );
rtDeclareVariable(Ray   , currentRay , rtCurrentRay          , );
rtDeclareVariable(float , tHit       , rtIntersectionDistance, );
rtDeclareVariable(NormalRayPayload, normalRayPayload, rtPayload, );
rtDeclareVariable(ShadowRayPayload, shadowRayPayload, rtPayload, );

rtDeclareVariable(Index, materialIndex, , );



/* 
 * ===  FUNCTION  ==============================================================
 *         Name:  handleNormalRayClosestHit
 *  Description:  
 * =============================================================================
 */
RT_PROGRAM void handleNormalRayClosestHit()
{
    normalRayPayload.isHit = true;

    Intersection & intersection = normalRayPayload.intersection;
    DifferentialGeometry * dg = intersection.dg();

    *dg = geometricDG;
//    if (launchIndex.x == 449 && launchIndex.y == 252) {
//        rtPrintf("before\n");
//        rtPrintf("point "); dump(dg->point); rtPrintf("\n");
//        rtPrintf("normal "); dump(dg->normal); rtPrintf("\n");
//    }
    dg->point  = rtTransformPoint(RT_OBJECT_TO_WORLD, dg->point);
    dg->normal = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, dg->normal));
    dg->normal = faceforward(dg->normal, -currentRay.direction, dg->normal);
//    if (launchIndex.x == 449 && launchIndex.y == 252) {
//        rtPrintf("after\n");
//        rtPrintf("point "); dump(dg->point); rtPrintf("\n");
//        rtPrintf("normal "); dump(dg->normal); rtPrintf("\n");
//    }

    const float3 & nn = dg->normal;
    const float3 & sn = normalize(dg->dpdu);
    const float3 tn = cross(nn, sn);
//    if (launchIndex.x == 449 && launchIndex.y == 252) {
//        rtPrintf("sn "); dump(sn); rtPrintf("\n");
//        rtPrintf("tn "); dump(tn); rtPrintf("\n");
//        rtPrintf("nn "); dump(nn); rtPrintf("\n");
//    }
    float * w2o = intersection.worldToObject()->getData();
    w2o[ 0] = sn.x; w2o[ 1] = sn.y; w2o[ 2] = sn.z; w2o[ 3] = 0.0f;
    w2o[ 4] = tn.x; w2o[ 5] = tn.y; w2o[ 6] = tn.z; w2o[ 7] = 0.0f;
    w2o[ 8] = nn.x; w2o[ 9] = nn.y; w2o[10] = nn.z; w2o[11] = 0.0f;
    w2o[12] = 0.0f; w2o[13] = 0.0f; w2o[14] = 0.0f; w2o[15] = 1.0f;

    // BSDF
    Matte & material = GET_MATERIAL(Matte, materialIndex);
    BSDF * bsdf = intersection.bsdf();
    *bsdf = BSDF(*dg, geometricDG.normal);

    // BxDFs
    /*TODO*/
    bsdf->m_nBxDFs = 1;
    Lambertian * bxdf = reinterpret_cast<Lambertian *>(bsdf->bxdfList());
    *bxdf = Lambertian(material.m_kd);
}   /* -----  end of function handleNormalRayClosestHit  ----- */
