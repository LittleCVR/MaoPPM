#include "hip/hip_runtime.h"
/*
 * =============================================================================
 *
 *       Filename:  Glass.cu
 *
 *    Description:  
 *
 *        Version:  1.0
 *        Created:  2011-06-28 22:13:53
 *
 *         Author:  Chun-Wei Huang (LittleCVR), 
 *        Company:  Communication & Multimedia Laboratory,
 *                  Department of Computer Science & Information Engineering,
 *                  National Taiwan University
 *
 * =============================================================================
 */

#include    "Glass.h"

/*----------------------------------------------------------------------------
 *  Header files from OptiX
 *----------------------------------------------------------------------------*/
#include    <optix_world.h>

/*----------------------------------------------------------------------------
 *  header files of our own
 *----------------------------------------------------------------------------*/
#include    "global.h"
#include    "payload.h"
#include    "utility.h"
#include    "BSDF.h"
#include    "DifferentialGeometry.h"
#include    "Intersection.h"

/*----------------------------------------------------------------------------
 *  namespace
 *----------------------------------------------------------------------------*/
using namespace optix;
using namespace MaoPPM;



rtDeclareVariable(uint2, launchIndex, rtLaunchIndex, );
rtDeclareVariable(uint2, launchSize ,              , );

rtBuffer<float,  1>  sampleList;

rtDeclareVariable(rtObject, rootObject, , );
rtDeclareVariable(DifferentialGeometry, geometricDG, attribute differential_geometry, ); 

rtDeclareVariable(float , rayEpsilon ,                       , );
rtDeclareVariable(Ray   , currentRay , rtCurrentRay          , );
rtDeclareVariable(float , tHit       , rtIntersectionDistance, );
rtDeclareVariable(NormalRayPayload, normalRayPayload, rtPayload, );
rtDeclareVariable(ShadowRayPayload, shadowRayPayload, rtPayload, );

rtDeclareVariable(Index, materialIndex, , );



/* 
 * ===  FUNCTION  ==============================================================
 *         Name:  handleNormalRayClosestHit
 *  Description:  
 * =============================================================================
 */
RT_PROGRAM void handleNormalRayClosestHit()
{
    normalRayPayload.isHit = true;
    // Intersection
    Intersection * intersection = normalRayPayload.intersection();
    // Differential geometry.
    *(intersection->dg()) = geometricDG;
    // Material
    intersection->setMaterialPointer(GET_MATERIAL_POINTER(materialIndex));
}   /* -----  end of function handleNormalRayClosestHit  ----- */
