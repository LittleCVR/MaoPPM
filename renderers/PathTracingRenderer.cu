#include "hip/hip_runtime.h"
/*
 * =============================================================================
 *
 *       Filename:  PathTracingRenderer.cu
 *
 *    Description:  
 *
 *        Version:  1.0
 *        Created:  2011-06-24 23:20:49
 *
 *         Author:  Chun-Wei Huang (LittleCVR), 
 *        Company:  Communication & Multimedia Laboratory,
 *                  Department of Computer Science & Information Engineering,
 *                  National Taiwan University
 *
 * =============================================================================
 */

#include    "PathTracingRenderer.h"

/*----------------------------------------------------------------------------
 *  Header files from OptiX
 *----------------------------------------------------------------------------*/
#include    <optix_world.h>

/*----------------------------------------------------------------------------
 *  header files of our own
 *----------------------------------------------------------------------------*/
#include    "global.h"
#include    "payload.h"
#include    "utility.h"
#include    "BSDF.h"
#include    "Light.h"

/*----------------------------------------------------------------------------
 *  namespace
 *----------------------------------------------------------------------------*/
using namespace optix;
using namespace MaoPPM;



rtDeclareVariable(uint2, launchIndex, rtLaunchIndex, );
rtDeclareVariable(uint2, launchSize ,              , );

rtBuffer<float4,      2>  outputBuffer;
rtBuffer<float,       1>  sampleList;

rtDeclareVariable(uint, nSamplesPerThread  , , );
rtDeclareVariable(uint, maxRayDepth        , , );
rtDeclareVariable(uint, frameCount         , , );

rtDeclareVariable(float3, cameraPosition, , );
rtDeclareVariable(float3, cameraU       , , );
rtDeclareVariable(float3, cameraV       , , );
rtDeclareVariable(float3, cameraW       , , );

rtDeclareVariable(NormalRayPayload, normalRayPayload, rtPayload, );
rtDeclareVariable(ShadowRayPayload, shadowRayPayload, rtPayload, );



/* 
 * ===  FUNCTION  ==============================================================
 *         Name:  generatePixelSamples
 *  Description:  
 * =============================================================================
 */
RT_PROGRAM void trace()
{
    uint offset = LAUNCH_OFFSET_2D(launchIndex, launchSize);
    uint sampleIndex = nSamplesPerThread * offset;

    Ray ray;
    NormalRayPayload payload;
    uint depth = 0;
    float3 wo, wi, L = make_float3(0.0f), throughput = make_float3(1.0f);
    /* TODO */
    Intersection * intersection = NULL;
    BSDF bsdf;
    for (uint i = 0; i < maxRayDepth; ++i) {
        // Start from camera.
        if (depth == 0) {
            /* TODO: move this task to the camera class */
            // Generate camera ray.
            float2 screenSize = make_float2(outputBuffer.size());
            float2 sample = GET_2_SAMPLES(sampleList, sampleIndex);
            float2 cameraRayDirection = (make_float2(launchIndex) + sample) / screenSize * 2.0f - 1.0f;
            wi = normalize(cameraRayDirection.x*cameraU + cameraRayDirection.y*cameraV + cameraW);
            ray = Ray(cameraPosition, wi, NormalRay, rayEpsilon);
        }
        // Start from surface.
        else {
            /*TODO*/
            float  probability;
            float3 sample = GET_3_SAMPLES(sampleList, sampleIndex);
            float3 f = bsdf.sampleF(wo, &wi, sample, &probability);
            if (probability == 0.0f) continue;
            throughput = f * throughput * fabsf(dot(wi, intersection->dg()->normal)) / probability;
            ray = Ray(intersection->dg()->point, wi, NormalRay, rayEpsilon);
        }

        // trace ray
        payload.reset();
        rtTrace(rootObject, ray, payload);
        if (!payload.isHit) continue;

        ++depth;
        wo = -wi;
        intersection = payload.intersection();
        intersection->getBSDF(&bsdf);

        // Evaluate radiance.
        L += throughput * estimateAllDirectLighting(
                intersection->dg()->point, bsdf, wo);
    }

    if (frameCount == 0)
        outputBuffer[launchIndex] = make_float4(0.0f);
    float frame = static_cast<float>(frameCount);
    outputBuffer[launchIndex] = (1.0f / (frame + 1.0f)) * make_float4(L, 1.0f) +
        (frame / (frame + 1.0f)) * outputBuffer[launchIndex];
}   /* -----  end of function generatePixelSamples  ----- */
