#include "hip/hip_runtime.h"
/*
 * =============================================================================
 *
 *       Filename:  IGPPMRenderer.cu
 *
 *    Description:  The Importons Guided Progressive Photon Map Renderer.
 *
 *        Version:  1.0
 *        Created:  2011-06-21 08:02:37
 *
 *         Author:  Chun-Wei Huang (LittleCVR), 
 *        Company:  Communication & Multimedia Laboratory,
 *                  Department of Computer Science & Information Engineering,
 *                  National Taiwan University
 *
 * =============================================================================
 */

#include    "IGPPMRenderer.h"

/*----------------------------------------------------------------------------
 *  Header files from OptiX
 *----------------------------------------------------------------------------*/
#include    <optix_world.h>

/*----------------------------------------------------------------------------
 *  header files of our own
 *----------------------------------------------------------------------------*/
#include    "global.h"
#include    "payload.h"
#include    "utility.h"
#include    "BSDF.h"
#include    "Camera.h"
#include    "Light.h"

/*----------------------------------------------------------------------------
 *  namespace
 *----------------------------------------------------------------------------*/
using namespace optix;
using namespace MaoPPM;



typedef IGPPMRenderer::PixelSample  PixelSample;
typedef IGPPMRenderer::Importon     Importon;
#define Photon IGPPMRenderer::Photon



rtDeclareVariable(uint2, launchIndex, rtLaunchIndex, );
rtDeclareVariable(uint2, launchSize ,              , );

rtBuffer<float4,      2>  outputBuffer;
rtBuffer<PixelSample, 2>  pixelSampleList;
rtBuffer<Importon,    1>  importonList;
rtBuffer<Photon,      1>  photonList;
rtBuffer<Photon,      1>  photonMap;
rtBuffer<float,       1>  sampleList;

rtDeclareVariable(uint, frameCount         , , );
rtDeclareVariable(uint, guidedByImportons  , , );
rtDeclareVariable(uint, nSamplesPerThread  , , );
rtDeclareVariable(uint, nImportonsPerThread, , );
rtDeclareVariable(uint, nPhotonsPerThread  , , );
rtDeclareVariable(uint, nEmittedPhotons    , , );
rtDeclareVariable(uint, maxRayDepth        , , );

rtDeclareVariable(Camera, camera, , );

rtDeclareVariable(NormalRayPayload, normalRayPayload, rtPayload, );
rtDeclareVariable(ShadowRayPayload, shadowRayPayload, rtPayload, );



/* 
 * ===  FUNCTION  ==============================================================
 *         Name:  generatePixelSamples
 *  Description:  
 * =============================================================================
 */
RT_PROGRAM void generatePixelSamples()
{
    // Clear output buffer.
    if (frameCount == 0)
        outputBuffer[launchIndex] = make_float4(0.0f, 0.0f, 0.0f, 1.0f);
    // Clear pixel sample.
    PixelSample & pixelSample = pixelSampleList[launchIndex];
    pixelSample.reset();

    // Generate camera ray.
    unsigned int offset = LAUNCH_OFFSET_2D(launchIndex, launchSize);
    unsigned int sampleIndex = nSamplesPerThread * offset;
    float2 sample = GET_2_SAMPLES(sampleList, sampleIndex);
    Ray ray = camera.generateCameraRay(
            launchIndex.x, launchIndex.y, sample, NormalRay, rayEpsilon);

    // Allocate memory for intersection.
    Intersection * intersection = LOCAL_HEAP_GET_OBJECT_POINTER(Intersection,
            LOCAL_HEAP_GET_CURRENT_INDEX() + sizeof(Intersection) * offset);
    // Trace until non-specular surface.
    BSDF bsdf;
    uint depth = 0;
    if (!traceUntilNonSpecularSurface(&ray, maxRayDepth, &depth,
                intersection, &bsdf, &pixelSample.wo, &pixelSample.throughput))
    {
        return;
    }
    pixelSample.flags |= PixelSample::isHit;
    pixelSample.setIntersection(intersection);
}   /* -----  end of function generatePixelSamples  ----- */



/* 
 * ===  FUNCTION  ==============================================================
 *         Name:  shootImportons
 *  Description:  
 * =============================================================================
 */
RT_PROGRAM void shootImportons()
{
    PixelSample & pixelSample = pixelSampleList[launchIndex];
    // Does not have to shoot importons if pixel sample was not hit.
    if (!(pixelSample.flags & PixelSample::isHit))
        return;
    if (frameCount != 0 && !(pixelSample.flags & PixelSample::Regather))
        return;
    pixelSample.flags &= ~PixelSample::Regather;
    pixelSample.nEmittedPhotonsOffset = nEmittedPhotons;

    // Prepare offset variables.
    unsigned int offset = LAUNCH_OFFSET_2D(launchIndex, launchSize);
    unsigned int sampleIndex   = offset * nSamplesPerThread;
    unsigned int importonIndex = offset * nImportonsPerThread;

    for (uint i = 0; i < nImportonsPerThread; i++)
        importonList[importonIndex+i].reset();

    const Intersection * pIntersection = pixelSample.intersection();
    const BSDF pBSDF = pIntersection->getBSDF();

    // other importons
    for (uint i = 0; i < nImportonsPerThread; i++) {
        Importon & importon = importonList[importonIndex+i];
        // Allocate memory for intersection. This is for importon.
        Intersection * intersection = LOCAL_HEAP_GET_OBJECT_POINTER(Intersection,
                LOCAL_HEAP_GET_CURRENT_INDEX() + (offset + i) * sizeof(Intersection));
        BSDF bsdf;

        float probability;
        float3 sample = GET_3_SAMPLES(sampleList, sampleIndex);
        Ray ray(camera.position, -pixelSample.wo, NormalRay, rayEpsilon);
        if (!bounce(&ray, *pIntersection->dg(), pBSDF, sample, &probability, &importon.throughput))
            continue;

        uint depth = 1;
        if (!traceUntilNonSpecularSurface(&ray, maxRayDepth, &depth,
                    intersection, &bsdf, &importon.wo, &importon.throughput))
        {
            continue;
        }

        importon.flags |= Importon::isHit;
        importon.setIntersection(intersection);

//        /* TODO */
//        float3 position = intersection->dg()->point;
//        float3 pos = transformPoint(camera.worldToRaster(), position);
//        uint2  ras = make_uint2(pos.x, pos.y);
//        if (ras.x < camera.width && ras.y < camera.height) {
//            if (isVisible(camera.position, position))
//                outputBuffer[ras] += make_float4(0.5f, 0.0f, 0.0f, 0.0f);
//        }
    }
}   /* -----  end of function shootImportons  ----- */



/* 
 * ===  FUNCTION  ==============================================================
 *         Name:  shootPhotons
 *  Description:  
 * =============================================================================
 */
RT_PROGRAM void shootPhotons()
{
    unsigned int offset = LAUNCH_OFFSET_2D(launchIndex, launchSize);
    unsigned int sampleIndex = nSamplesPerThread * offset;
    unsigned int photonIndex = nPhotonsPerThread * offset;

    // Clear photon list.
    for (uint i = 0; i < nPhotonsPerThread; i++)
        photonList[photonIndex+i].reset();

    // Allocate memory for intersection.
    Intersection * intersection = LOCAL_HEAP_GET_OBJECT_POINTER(Intersection,
            LOCAL_HEAP_GET_CURRENT_INDEX() + offset * sizeof(Intersection));

    Ray ray;
    BSDF bsdf;
    uint depth = 0;
    float3 wo, wi, flux;
    bool isCausticPhoton = true;
    unsigned int binFlags = 0;
    for (uint i = 0; i < nPhotonsPerThread; i++) {
        // starts from lights
        if (depth == 0) {
            // sample light
            float lightSample = GET_1_SAMPLE(sampleList, sampleIndex);
            const Light * light = sampleOneLightUniformly(lightSample);
            // sample direction
            float  probability;
            float2 sample = GET_2_SAMPLES(sampleList, sampleIndex);
            /* TODO: add guidedByImportons */
            float3 Le = light->sampleL(sample, &wo, &probability);
            flux = Le / probability;
            ray = Ray(light->position, wo, NormalRay, rayEpsilon);
//            unsigned int thetaBin, phiBin;
//            if (!guidedByImportons) {
//                float2 s = make_float2(sample);
//                wo = sampleUniformSphere(s);
//                float theta = acosf(wo.z);
//                float phi   = acosf(wo.x);
//                if (wo.y < 0.0f) phi += M_PIf;
//                thetaBin = fminf(N_THETA-1,
//                        floorf(theta / M_PIf * static_cast<float>(N_THETA)));
//                phiBin = fminf(N_PHI-1,
//                        floorf(phi / (2.0f*M_PIf) * static_cast<float>(N_PHI)));
//                flux = light.flux * 4.0f * M_PIf;
//            } else {
//                // CDF
//                uint index = 0;
//                for (uint j = 0; j < N_THETA*N_PHI; ++j)
//                    if (sample.z <= light.cdf[j]) {
//                        index = j;
//                        break;
//                    }
//                thetaBin = index / N_PHI;
//                phiBin   = index % N_PHI;
//                float zMax = static_cast<float>(thetaBin+0) / N_THETA;
//                float zMin = static_cast<float>(thetaBin+1) / N_THETA;
//                float pMax = static_cast<float>(phiBin+0) * (2.0f * M_PIf) / N_PHI;
//                float pMin = static_cast<float>(phiBin+1) * (2.0f * M_PIf) / N_PHI;
//                float2 s = make_float2(sample);
//                s.x = s.x * (zMax-zMin) + zMin;
//                s.y = (s.y * (pMax-pMin) + pMin) / (2.0f * M_PIf);
//                wo = sampleUniformSphere(s);
//                flux = light.flux * 4.0f * M_PIf * light.normalizedArea(thetaBin, phiBin) /
//                    (index == 0 ? light.cdf[index] : (light.cdf[index]-light.cdf[index-1]));
//                if (launchIndex.x == 128 && launchIndex.y == 128) {
//                    float theta = acosf(wo.z);
//                    float phi   = acosf(wo.x);
//                    if (wo.y < 0.0f) phi += M_PIf;
//                    theta = theta * 180.0f / M_PIf;
//                    phi   = phi   * 180.0f / M_PIf;
//                    rtPrintf("tb: %u, pb: %u, zMin: %f, zMax: %f, pMin: %f, pMax: %f, ",
//                            thetaBin, phiBin, zMin, zMax, pMin, pMax);
//                    rtPrintf("s.x: %f, s.y: %f, theta: %f, phi: %f, flux: %f %f %f\n",
//                            s.x, s.y, theta, phi, flux.x, flux.y, flux.z);
//                }
//            }
//            binFlags = (thetaBin << 24) | (phiBin << 16);
        }
        // starts from surface
        else {
            float  probability;
            float3 sample = GET_3_SAMPLES(sampleList, sampleIndex);
            // remember that we are now shooting rays from a light
            // thus wo and wi must be swapped
            if (!bounce(&ray, *intersection->dg(), bsdf, sample, &probability, &flux))
                continue;
        }

        // trace ray
        if (!traceUntilNonSpecularSurface(&ray, maxRayDepth, &depth,
                    intersection, &bsdf, &wo, &flux))
        {
            continue;
        }
        wi = wo;

        // create photon
        Photon & photon = photonList[photonIndex+i];
        photon.flags |= binFlags;
        if (depth == 1)
            photon.flags |= Photon::Direct;
        else if (isCausticPhoton)
            photon.flags |= Photon::Caustic;
        else
            photon.flags |= Photon::Indirect;
        photon.position = intersection->dg()->point;
        photon.wi       = wi;
        photon.flux     = flux;

//        /* TODO */
//        if (photon.flags & Photon::Direct) {
//            float3 position = intersection->dg()->point;
//            float3 pos = transformPoint(camera.worldToRaster(), position);
//            uint2  ras = make_uint2(pos.x, pos.y);
//            if (ras.x < camera.width && ras.y < camera.height) {
//                if (isVisible(camera.position, position))
//                    outputBuffer[ras] += make_float4(0.5f, 0.0f, 0.0f, 0.0f);
//            }
//        }

        // After traceUntilNonSpecularSurface(),
        // photons should be all indirect now.
        isCausticPhoton = false;
        // Reset depth if necessary.
        if (depth % maxRayDepth == 0) {
            depth = 0;
            isCausticPhoton = true;
        }
    }
}   /* -----  end of function shootPhotons  ----- */



/* 
 * ===  FUNCTION  ==============================================================
 *         Name:  gatherPhotons
 *  Description:  
 * =============================================================================
 */
RT_PROGRAM void gatherPhotons()
{
    // Do not have to gather photons if pixel sample was not hit.
    PixelSample & pixelSample = pixelSampleList[launchIndex];
    if (!(pixelSample.flags & PixelSample::isHit)) return;

    uint offset = LAUNCH_OFFSET_2D(launchIndex, launchSize);
    uint importonIndex = nImportonsPerThread * offset;

    // Evaluate direct illumination.
    Intersection * intersection = pixelSample.intersection();
    BSDF bsdf; intersection->getBSDF(&bsdf);
    float3 direct = pixelSample.throughput *
        estimateAllDirectLighting(intersection->dg()->point, bsdf, pixelSample.wo);

    GatheredPhoton * gatheredPhotonList = NULL;
    if (frameCount == 0) {
        Index gatheredPhotonListIndex = LOCAL_HEAP_GET_CURRENT_INDEX() +
            offset * nPhotonsUsed * sizeof(GatheredPhoton);
        gatheredPhotonList =
            LOCAL_HEAP_GET_OBJECT_POINTER(GatheredPhoton, gatheredPhotonListIndex);
    }

    // Gather pixel sample first.
    uint nAccumulatedPhotons = 0;
    float3 flux = make_float3(0.0f);
    float maxDistanceSquared = pixelSample.radiusSquared;
//    // First time we should use LimitedPhotonGatherer to find initial radius.
//    // Otherwise just gather all the photons in range.
//    if (frameCount == 0) {
//        flux = LimitedPhotonGatherer::accumulateFlux(
//                intersection->dg()->point, pixelSample.wo, &bsdf,
//                &photonMap[0], &maxDistanceSquared, &nAccumulatedPhotons,
//                gatheredPhotonList, Photon::Flag(Photon::Caustic));
//        // maxDistanceSquared may be shrinked. So write it back to pixelSample.
//        pixelSample.radiusSquared = maxDistanceSquared;
//    }
//    else {  // frameCount != 0
//        flux = PhotonGatherer::accumulateFlux(
//                intersection->dg()->point, pixelSample.wo, &bsdf,
//                &photonMap[0], &maxDistanceSquared, &nAccumulatedPhotons,
//                Photon::Flag(Photon::Caustic));
//    }
//    pixelSample.shrinkRadius(flux, nAccumulatedPhotons);
//
//    // Caustic.
//    float3 caustic = pixelSample.throughput *
//        pixelSample.flux / (M_PIf * pixelSample.radiusSquared) /
//        nEmittedPhotons;

    // Compute indirect illumination.
    float greatestReductionFactor2 = 0.0f;
    for (uint i = 0; i < nImportonsPerThread; ++i) {
        Importon & importon = importonList[importonIndex+i];
        if (importon.flags & Importon::isHit) {
            intersection = importon.intersection();
            intersection->getBSDF(&bsdf);
            maxDistanceSquared = importon.radiusSquared;
            // First time we should use LimitedPhotonGatherer to find initial radius.
            // Otherwise just gather all the photons in range.
            if (frameCount == 0) {
                flux = LimitedPhotonGatherer::accumulateFlux(
                        intersection->dg()->point, importon.wo, &bsdf,
                        &photonMap[0], &maxDistanceSquared, &nAccumulatedPhotons,
                        gatheredPhotonList, Photon::Flag(Photon::All));
                // KdTree::find() may shrink the radius. So write it back to pixelSample.
                importon.radiusSquared = maxDistanceSquared;
            }
            else {  // frameCount != 0
                flux = PhotonGatherer::accumulateFlux(
                        intersection->dg()->point, importon.wo, &bsdf,
                        &photonMap[0], &maxDistanceSquared, &nAccumulatedPhotons,
                        Photon::Flag(Photon::All),
                        &lightList[0], dot(importon.throughput, importon.throughput));
            }

            float reductionFactor2;
            importon.shrinkRadius(flux, nAccumulatedPhotons, &reductionFactor2);
            if (greatestReductionFactor2 < reductionFactor2)
                greatestReductionFactor2 = reductionFactor2;
        }
    }

    // Indirect
    float3 indirect = make_float3(0.0f);
    unsigned int nValidImportons = 0;
    for (uint i = 0; i < nImportonsPerThread; i++) {
        Importon & importon = importonList[importonIndex+i];
        if (importon.flags & Importon::isHit) {
            ++nValidImportons;
            float3 Li = importon.flux / (M_PIf * importon.radiusSquared);
            indirect += importon.throughput * Li;
        }
    }
    if (nValidImportons != 0) {
        indirect *= pixelSample.throughput /
            (nEmittedPhotons - pixelSample.nEmittedPhotonsOffset) / nValidImportons;
    }

    /* TODO: test */
    if (nValidImportons == 0 || greatestReductionFactor2 > 0.95f)
        pixelSample.flags |= PixelSample::Regather;

    // Average.
    float nGathered = static_cast<float>(pixelSample.nGathered);
    indirect = (1.0f / (nGathered + 1.0f)) * indirect +
        (nGathered / (nGathered + 1.0f)) * pixelSample.indirect;
    if (pixelSample.flags & PixelSample::Regather) {
        pixelSample.indirect = indirect;
        ++pixelSample.nGathered;
    }
    outputBuffer[launchIndex] = make_float4(direct + indirect, 1.0f);
}   /* -----  end of function gatherPhotons  ----- */
