#include "hip/hip_runtime.h"
/*
 * =============================================================================
 *
 *       Filename:  PPMRenderer.cu
 *
 *    Description:  
 *
 *        Version:  1.0
 *        Created:  2011-06-27 11:09:25
 *
 *         Author:  Chun-Wei Huang (LittleCVR), 
 *        Company:  Communication & Multimedia Laboratory,
 *                  Department of Computer Science & Information Engineering,
 *                  National Taiwan University
 *
 * =============================================================================
 */

#include    "PPMRenderer.h"

/*----------------------------------------------------------------------------
 *  Header files from OptiX
 *----------------------------------------------------------------------------*/
#include    <optix_world.h>

/*----------------------------------------------------------------------------
 *  header files of our own
 *----------------------------------------------------------------------------*/
#include    "global.h"
#include    "payload.h"
#include    "utility.h"
#include    "BSDF.h"
#include    "Camera.h"
#include    "Light.h"

/*----------------------------------------------------------------------------
 *  namespace
 *----------------------------------------------------------------------------*/
using namespace optix;
using namespace MaoPPM;



typedef PPMRenderer::PixelSample  PixelSample;
typedef MaoPPM::Photon            Photon;



rtDeclareVariable(uint2, launchIndex, rtLaunchIndex, );
rtDeclareVariable(uint2, launchSize ,              , );

rtBuffer<float4,      2>  outputBuffer;
rtBuffer<PixelSample, 2>  pixelSampleList;
rtBuffer<Photon,      1>  photonList;
rtBuffer<Photon,      1>  photonMap;
rtBuffer<float,       1>  sampleList;

rtDeclareVariable(uint, frameCount         , , );
rtDeclareVariable(uint, nSamplesPerThread  , , );
rtDeclareVariable(uint, nPhotonsPerThread  , , );
rtDeclareVariable(uint, nEmittedPhotons    , , );
rtDeclareVariable(uint, maxRayDepth        , , );

rtDeclareVariable(Camera, camera, , );

rtDeclareVariable(NormalRayPayload, normalRayPayload, rtPayload, );
rtDeclareVariable(ShadowRayPayload, shadowRayPayload, rtPayload, );



/* 
 * ===  FUNCTION  ==============================================================
 *         Name:  generatePixelSamples
 *  Description:  
 * =============================================================================
 */
RT_PROGRAM void generatePixelSamples()
{
    // Clear output buffer.
    if (frameCount == 0)
        outputBuffer[launchIndex] = make_float4(0.0f, 0.0f, 0.0f, 1.0f);
    // Clear pixel sample.
    PixelSample & pixelSample = pixelSampleList[launchIndex];
    pixelSample.reset();

    // Generate camera ray.
    unsigned int offset = LAUNCH_OFFSET_2D(launchIndex, launchSize);
    unsigned int sampleIndex = nSamplesPerThread * offset;
    float2 sample = GET_2_SAMPLES(sampleList, sampleIndex);
    Ray ray = camera.generateCameraRay(
            launchIndex.x, launchIndex.y, sample, NormalRay, rayEpsilon);

    // Allocate memory for intersection.
    Intersection * intersection = LOCAL_HEAP_GET_OBJECT_POINTER(Intersection,
            LOCAL_HEAP_GET_CURRENT_INDEX() + sizeof(Intersection) * offset);
    // Trace until non-specular surface.
    BSDF bsdf;
    uint depth = 0;
    if (!traceUntilNonSpecularSurface(&ray, maxRayDepth, &depth,
                intersection, &bsdf, &pixelSample.wo, &pixelSample.throughput))
    {
        return;
    }
    pixelSample.flags |= PixelSample::isHit;
    pixelSample.setIntersection(intersection);
}   /* -----  end of function generatePixelSamples  ----- */



/* 
 * ===  FUNCTION  ==============================================================
 *         Name:  shootPhotons
 *  Description:  
 * =============================================================================
 */
RT_PROGRAM void shootPhotons()
{
    unsigned int offset = LAUNCH_OFFSET_2D(launchIndex, launchSize);
    unsigned int sampleIndex = nSamplesPerThread * offset;
    unsigned int photonIndex = nPhotonsPerThread * offset;

    // Clear photon list.
    for (uint i = 0; i < nPhotonsPerThread; i++)
        photonList[photonIndex+i].reset();

    // Allocate memory for intersection.
    Intersection * intersection = LOCAL_HEAP_GET_OBJECT_POINTER(Intersection,
            LOCAL_HEAP_GET_CURRENT_INDEX() + offset * sizeof(Intersection));

    // Shoot photons.
    Ray ray;
    BSDF bsdf;
    float3 wo, wi, flux;
    unsigned int depth = 0;
    for (unsigned int i = 0; i < nPhotonsPerThread; i++) {
        // starts from lights
        if (depth == 0) {
            // sample light
            float lightSample = GET_1_SAMPLE(sampleList, sampleIndex);
            const Light * light = sampleOneLightUniformly(lightSample);
            flux = light->flux * 4.0f * M_PIf;
            // sample direction
            float2 sample = GET_2_SAMPLES(sampleList, sampleIndex);
            wo = sampleUniformSphere(sample);
            ray = Ray(light->position, wo, NormalRay, rayEpsilon);
        }
        // starts from surface
        else {
            float  probability;
            float3 sample = GET_3_SAMPLES(sampleList, sampleIndex);
            if (!bounce(&ray, *intersection->dg(), bsdf, sample, &probability, &flux))
                continue;
        }

        // trace ray
        if (!traceUntilNonSpecularSurface(&ray, maxRayDepth, &depth,
                    intersection, &bsdf, &wo, &flux))
        {
            continue;
        }
        wi = wo;

        // create photon
        Photon & photon = photonList[photonIndex+i];
        if (depth == 1)
            photon.flags |= Photon::Direct;
        else
            photon.flags |= Photon::Indirect;
        photon.position = intersection->dg()->point;
        photon.wi       = wi;
        photon.flux     = flux;

//        /* TODO */
//        float3 pos = transformPoint(camera.worldToRaster(), photon.position);
//        uint2  ras = make_uint2(pos.x, pos.y);
//        if (ras.x < camera.width && ras.y < camera.height) {
//            if (isVisible(camera.position, photon.position))
//                outputBuffer[ras] += make_float4(0.5f, 0.0f, 0.0f, 0.0f);
//        }

        // Reset depth if necessary.
        if (depth % maxRayDepth == 0)
            depth = 0;
    }
}   /* -----  end of function shootPhotons  ----- */



/* 
 * ===  FUNCTION  ==============================================================
 *         Name:  
 *  Description:  
 * =============================================================================
 */
RT_PROGRAM void estimateDensity()
{
    // Do not have to gather photons if pixel sample was not hit.
    PixelSample & pixelSample = pixelSampleList[launchIndex];
    if (!(pixelSample.flags & PixelSample::isHit)) return;

    // Evaluate direct illumination.
    Intersection * intersection = pixelSample.intersection();
    BSDF bsdf; intersection->getBSDF(&bsdf);
    pixelSample.direct = pixelSample.throughput *
        estimateAllDirectLighting(intersection->dg()->point, bsdf, pixelSample.wo);

    // Gather.
    uint nAccumulatedPhotons = 0;
    float3 flux = make_float3(0.0f);
    float maxDistanceSquared = pixelSample.radiusSquared;
    // First time we should use LimitedPhotonGatherer to find initial radius.
    // Otherwise just gather all the photons in range.
    if (frameCount == 0) {
        unsigned int offset = LAUNCH_OFFSET_2D(launchIndex, launchSize);
        unsigned int gatheredPhotonListIndex = LOCAL_HEAP_GET_CURRENT_INDEX() +
            offset * nPhotonsUsed * sizeof(GatheredPhoton);
        GatheredPhoton * gatheredPhotonList =
            LOCAL_HEAP_GET_OBJECT_POINTER(GatheredPhoton, gatheredPhotonListIndex);
        flux = LimitedPhotonGatherer::accumulateFlux(
                intersection->dg()->point, pixelSample.wo, &bsdf,
                &photonMap[0], &maxDistanceSquared, &nAccumulatedPhotons,
                gatheredPhotonList);
        // maxDistanceSquared may be shrinked. So write it back to pixelSample.
        pixelSample.radiusSquared = maxDistanceSquared;
    }
    else {  // frameCount != 0
        flux = PhotonGatherer::accumulateFlux(
                intersection->dg()->point, pixelSample.wo, &bsdf,
                &photonMap[0], &maxDistanceSquared, &nAccumulatedPhotons);
    }
    // Finally shrink the radius.
    pixelSample.shrinkRadius(flux, nAccumulatedPhotons);

    // Output.
    float3 indirect = pixelSample.flux / (M_PIf * pixelSample.radiusSquared) / nEmittedPhotons;
    outputBuffer[launchIndex] = make_float4(pixelSample.direct + indirect, 1.0f);
}   /* -----  end of function gatherPhotons  ----- */
